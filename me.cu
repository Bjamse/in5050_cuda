#include "hip/hip_runtime.h"
#include <assert.h>
#include <errno.h>
#include <getopt.h>
#include <limits.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <inttypes.h>
#include <math.h>
#include <stdlib.h>

#include "me.h"
#include "tables.h"

__global__ void computeSADIntegrated(int left, int right, int top, int bottom, int w, int mx, int my, uint8_t *cu_offset, uint8_t *cu_ref, int *best_sad, int *best_x, int *best_y)
{
    int x = left + blockIdx.x * blockDim.x + threadIdx.x;
    int y = top + blockIdx.y * blockDim.y + threadIdx.y;

    if (x < right && y < bottom) {
        __shared__ int sad_shared[64];
        int i = threadIdx.y * blockDim.x + threadIdx.x;  // Thread index in the block

        if (i < 64) {
            int bx = i % 8;  // Block x index
            int by = i / 8;  // Block y index
            int index = (y + by) * w + (x + bx);
            sad_shared[i] = abs(cu_ref[index] - cu_offset[index]);
        }

        __syncthreads(); // Ensure all threads have written their SAD values

        // Reduction within the block
        int block_sad = 0;
        if (threadIdx.x == 0 && threadIdx.y == 0) {
            for (int j = 0; j < 64; ++j) {
                block_sad += sad_shared[j];
            }

            // Atomic update for best sad and corresponding coordinates
            if (atomicMin(best_sad, block_sad) == block_sad) {
                *best_x = x - mx;
                *best_y = y - my;
            }
        }
    }
}




/* Motion estimation for 8x8 block */
static void me_block_8x8(struct c63_common *cm, int mb_x, int mb_y,
    uint8_t *orig, uint8_t *ref, int color_component)
{
  struct macroblock *mb =
    &cm->curframe->mbs[color_component][mb_y*cm->padw[color_component]/8+mb_x];

  int range = cm->me_search_range;

  /* Quarter resolution for chroma channels. */
  if (color_component > 0) { range /= 2; }

  int left = mb_x * 8 - range;
  int top = mb_y * 8 - range;
  int right = mb_x * 8 + range;
  int bottom = mb_y * 8 + range;

  int w = cm->padw[color_component];
  int h = cm->padh[color_component];

  /* Make sure we are within bounds of reference frame. TODO: Support partial
     frame bounds. */
  if (left < 0) { left = 0; }
  if (top < 0) { top = 0; }
  if (right > (w - 8)) { right = w - 8; }
  if (bottom > (h - 8)) { bottom = h - 8; }

  int mx = mb_x * 8;
  int my = mb_y * 8;

  int best_sad = INT_MAX;


  uint8_t *cu_orig, *cu_ref;


  hipMemcpy(cu_orig, orig, w*h * sizeof(uint8_t), hipMemcpyHostToDevice);
  hipMemcpy(cu_ref, ref, w*h * sizeof(uint8_t), hipMemcpyHostToDevice);


  uint8_t * cu_offsett = cu_orig + my*w+mx;
  // Define variables for minimum SAD and motion vectors

  int best_x = 0, best_y = 0;

  // Allocate memory for best_sad, best_x, best_y on device
  int *d_best_sad, *d_best_x, *d_best_y;
  hipMalloc(&d_best_sad, sizeof(int));
  hipMalloc(&d_best_x, sizeof(int));
  hipMalloc(&d_best_y, sizeof(int));
  hipMemcpy(d_best_sad, &best_sad, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_best_x, &best_x, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_best_y, &best_y, sizeof(int), hipMemcpyHostToDevice);

  // Define grid and block sizes
  dim3 blocks((right - left + 15) / 16, (bottom - top + 15) / 16);
  dim3 threads(16, 16);

  // Launch the kernel
  computeSADIntegrated<<<blocks, threads>>>(left, right, top, bottom, w, mx, my, cu_offsett, cu_ref, d_best_sad, d_best_x, d_best_y);

  // Copy results back to host
  hipMemcpy(&best_sad, d_best_sad, sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(&best_x, d_best_x, sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(&best_y, d_best_y, sizeof(int), hipMemcpyDeviceToHost);

  // Clean up
  hipFree(d_best_sad);
  hipFree(d_best_x);
  hipFree(d_best_y);

  hipFree(cu_orig);
  hipFree(cu_ref);

  /* Here, there should be a threshold on SAD that checks if the motion vector
     is cheaper than intraprediction. We always assume MV to be beneficial */

  /* printf("Using motion vector (%d, %d) with SAD %d\n", mb->mv_x, mb->mv_y,
     best_sad); */

  mb->use_mv = 1;
}





void c63_motion_estimate(struct c63_common *cm)
{
  /* Compare this frame with previous reconstructed frame */
  int mb_x, mb_y;

  /* Luma */
  for (mb_y = 0; mb_y < cm->mb_rows; ++mb_y)
  {
    for (mb_x = 0; mb_x < cm->mb_cols; ++mb_x)
    {
      me_block_8x8(cm, mb_x, mb_y, cm->curframe->orig->Y,
          cm->refframe->recons->Y, Y_COMPONENT);
    }
  }

  /* Chroma */
  for (mb_y = 0; mb_y < cm->mb_rows / 2; ++mb_y)
  {
    for (mb_x = 0; mb_x < cm->mb_cols / 2; ++mb_x)
    {
      me_block_8x8(cm, mb_x, mb_y, cm->curframe->orig->U,
          cm->refframe->recons->U, U_COMPONENT);
      me_block_8x8(cm, mb_x, mb_y, cm->curframe->orig->V,
          cm->refframe->recons->V, V_COMPONENT);
    }
  }
}



__global__ void cuda_mc_block_8x8(macroblock mb, int w, uint8_t *predicted, uint8_t *ref, int mv_x, int mv_y)
{
    if (!mb.use_mv) { return; }

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if within bounds to prevent out-of-bounds access
    if (x < 8 && y < 8) {
        predicted[y * w + x] = ref[(y + mv_y) * w + (x + mv_x)];
    }
}



// this had no impact on perfomance... its probably jsut too slow to move the data over... but i tried anyway, and the code looks almost nice
void c63_motion_compensate(struct c63_common *cm)
{
    int mb_x, mb_y;

    // Loop over all macroblocks
    for (mb_y = 0; mb_y < cm->mb_rows; ++mb_y)
    {
        for (mb_x = 0; mb_x < cm->mb_cols; ++mb_x)
        {
            // Handle Y, U, and V components separately due to different pointers
            int colors[] = {Y_COMPONENT, U_COMPONENT, V_COMPONENT};
            uint8_t* predicted_ptrs[] = {cm->curframe->predicted->Y, cm->curframe->predicted->U, cm->curframe->predicted->V};
            uint8_t* ref_ptrs[] = {cm->refframe->recons->Y, cm->refframe->recons->U, cm->refframe->recons->V};

            for (int color_idx = 0; color_idx < 3; ++color_idx)
            {
                int color = colors[color_idx];
                int w = cm->padw[color];
                macroblock *mb = &cm->curframe->mbs[color][mb_y * (w/8) + mb_x];
                uint8_t *predicted = predicted_ptrs[color_idx];
                uint8_t *ref = ref_ptrs[color_idx];

                int mv_x = mb->mv_x;
                int mv_y = mb->mv_y;

                // Ensure width and height limits are respected
                if (mb_x < w && mb_y < cm->padh[color])
                {
                    // Launch kernel for the current macroblock and color component
                    cuda_mc_block_8x8<<<1, dim3(8, 8)>>>(*mb, w, predicted, ref, mv_x, mv_y);
                }
            }
        }
    }

    // Synchronize after all kernel launches
    hipDeviceSynchronize();
}
