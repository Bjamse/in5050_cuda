#include "hip/hip_runtime.h"
#include <inttypes.h>
#include <math.h>
#include <stdlib.h>

#include "dsp.h"
#include "tables.h"

void sad_block_8x8(uint8_t *block1, uint8_t *block2, int stride, int *result)
{
  int u, v;

  *result = 0;

  for (v = 0; v < 8; ++v)
  {
    for (u = 0; u < 8; ++u)
    {
      *result += abs(block2[v*stride+u] - block1[v*stride+u]);
    }
  }
}
